#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T  0.01
#define kappa 0.000045
#define MAX_THREADS 512

__global__ void run_ftcs(double *u, double *prev, double k, double t, double x, long int n)
{
	long int i =  blockDim.x * blockIdx.x + threadIdx.x;
	if(i>0 && i<(n-1))
		u[i] = prev[i] + k*t/(x*x)*(prev[i-1]-2*prev[i]+prev[i+1]);
}

__global__ void borders(double *u, long int n)
{
	long int i =  blockDim.x * blockIdx.x + threadIdx.x;
	if(i==1) u[0] = u[n] = 0.0;
}


int main(void) 
{

    /********* HOST VARIABLES *********/
    double *tmp, *u, *u_prev;
    double x = 0, t = 0;
    long int i, n, maxloc = 0, blocks;

    n = tam/dx;
    u = malloc((n+1)*sizeof(double));
    u_prev = malloc((n+1)*sizeof(double));
    blocks = ceil((n+1)/MAX_THREADS);

    /********* CUDA VARIABLES *********/
    double *d_u, *d_u_prev;

    hipMalloc((void **)&d_u, n + 1);
	hipMalloc((void **)&d_u_prev, n + 1);
    hipMemcpy(d_u_prev, u_prev, n + 1, hipMemcpyHostToDevice);

    
    /********* FILLING U_PREV *********/
  
    for (i = 0; i < n + 1; i++) {
        if (x <= 0.5) u_prev[i] = 200*x;
        else        u_prev[i] = 200*(1.-x);
        x += dx;
    }

  

    
    while (t < T) {
        x = dx;
        run_ftcs<<<nblocks,THREADS_PER_BLOCK>>>(d_u, d_u_prev, kappa, dt, dx, n+1);
        hipDeviceSynchronize();
        borders<<<nblocks,THREADS_PER_BLOCK>>>(u_u, n);

        tmp = u_prev; 
        u_prev = u;
        u = tmp; /* troca entre ponteiros */
        t += dt;
    }

    hipMemcpy(u, d_u, n + 1, hipMemcpyDeviceToHost);

  /* Calculando o maior valor e sua localizacao */
 
    for (i = 1; i < n + 1; i++) 
        if (u[i] > u[maxloc]) maxloc = i;
    
    printf("Maior valor u[%ld] = %g\n", maxloc, u[maxloc]);


    free(u);
    free(u_prev);
    hipFree(u_u);
    hipFree(u_prev_prev);

    return 0;

}
